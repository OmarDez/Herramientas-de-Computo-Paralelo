#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <iostream>

using namespace std;

__global__ void MultiplicarMatricesSecuencial(float *matriz1_GPU, float *matriz2_GPU, float *matriz3_GPU, int TDM, size_t pitch){ //Este modulo usa 1 solo thread
    for(int i =0; i < TDM; i++){
        for(int j = 0; j < TDM; j++){
            float *elementos_matriz1 = (float *) ((char*)matriz1_GPU + j * pitch);                      //Obtenemos el j-esimo renglon de la matriz
            float *elementos_matriz3 = (float *) ((char*)matriz3_GPU + j * pitch);
            elementos_matriz3[i] = 0;
            for(int x = 0; x < TDM; x++){
                float *elementos_matriz2 = (float *) ((char*)matriz2_GPU + (x) * pitch);                
                elementos_matriz3[i] += elementos_matriz1[x] * elementos_matriz2[i];                    //Sumamos en la i-esima laumna del renglon de la matriz que trabajamos
                free(elementos_matriz2);
                
            }
        }
    }
}


__global__ void MultiplicarMatricesOn(float *matriz1_GPU, float *matriz2_GPU, float *matriz3_GPU, int TDM,size_t pitch){  //Este modulo usa n^2 threads
    const unsigned int idx = threadIdx.x + (blockDim.x *  blockIdx.x);
    
    const unsigned int j = idx / TDM;
    const unsigned int i = idx % TDM;
    
    float *elementos_matriz1 = (float *) ((char*)matriz1_GPU + j * pitch);
    float *elementos_matriz3 = (float *) ((char*)matriz3_GPU + j * pitch);
    elementos_matriz3[i] = 0;
    for(int x = 0; x < TDM; x ++){
        float *elementos_matriz2 = (float *) ((char*)matriz2_GPU + x * pitch); 
        elementos_matriz3[i] += elementos_matriz1[x] * elementos_matriz2[i];
        free(elementos_matriz2);
    }
}


int main(){
    int TDM = 50;
    
    int TDM2 = 1;
    unsigned int NDH = pow(TDM2,2);                                             // Número de hilos  
    unsigned int numero_bloques =  ceil( (float) NDH / (float) TDM2);           // Tamño de la matriz (cuadrada)TDM ); 
    unsigned int hilos_bloque = ceil( (float) NDH / (float) numero_bloques);    // Tamño de la matriz (cuadrada)ero_bloques );

    float matriz1_host[TDM][TDM];                                               //Creamos arrglos bidimensionales en la CPU
    float matriz2_host[TDM][TDM];

    for(int i = 0; i < TDM; i++){
        for(int j = 0; j < TDM; j++){
            matriz1_host[i][j] = (int)(i + j);                                  //Llenamos la matriz que creamos antes con valores
            matriz2_host[i][j] = (int)(i + j);
        }   
    }
    /* ********** Muestra las matrices que se van a multiplicar
    cout << "Matrices a multiplicar \nMatriz 1" << endl;
    for(int i = 0; i < TDM; i++){
        for(int j = 0; j < TDM; j++){
            cout << *(*(matriz1_host + i) + j) << "\t";
        }
        cout << "\n";
    }
    cout << "\nMatriz 2" << endl;
    for(int i = 0; i < TDM; i++){
        for(int j = 0; j < TDM; j++){
            cout << *(*(matriz2_host + i) + j) << '\t';
        }
        cout << "\n";
    }
    */
    size_t pitch;                                                                                       //  Esta variable contiene el tamaño de los espacios 
                                                                                                        //interiores de la matriz
    float *matriz1_GPU; hipMallocPitch(&matriz1_GPU, &pitch, TDM * sizeof(float), TDM );               //  Hacemos las reservaciones en memoria de las 
    float *matriz2_GPU; hipMallocPitch(&matriz2_GPU, &pitch, TDM * sizeof(float), TDM );               //matrices que vamos a necesitar
    float *matriz3_GPU; hipMallocPitch(&matriz3_GPU, &pitch, TDM * sizeof(float), TDM );

    hipMemcpy2D(matriz1_GPU, pitch, matriz1_host, TDM * sizeof(float), TDM * sizeof(float), TDM, hipMemcpyHostToDevice); //Copiamos los valores de la matriz en
    hipMemcpy2D(matriz2_GPU, pitch, matriz2_host, TDM * sizeof(float), TDM * sizeof(float), TDM, hipMemcpyHostToDevice); //en la CPU a la GPU

    hipEvent_t inicio, alto;                                                                               //Variables para el control de los eventos
    float tiempo_computo;                                                                                   // Varibale para almacenar el tiempo trancurrido (ms)

    for(TDM2 = 1; TDM2 <= TDM; TDM2++){
        
        NDH = pow(TDM2,2);                                                                                  //numero de hilos que se lanzarán
        numero_bloques =  ceil( (float) NDH / (float) TDM2);
        hilos_bloque = ceil( (float) NDH / (float) numero_bloques);
        tiempo_computo = 0;                                                                                 //Esta variable contendra el tiempo en ms que demora el evento
        hipEventCreate(&inicio); hipEventCreate(&alto);                                                   //Creamos los eventos
        hipEventRecord(inicio);                                                                            //Creamos una marca temporal, una especia de bandera 
        MultiplicarMatricesOn<<<numero_bloques, hilos_bloque>>>(matriz1_GPU, matriz2_GPU, matriz3_GPU, TDM2,  pitch);
        hipEventRecord(alto);                                                                              // Creamos una marca temporal, otra bandera
        hipEventSynchronize(alto);                                                                         // Bloquea la CPU para evitar que se continue con el programa hasta que se completen los eventos
        hipEventElapsedTime(&tiempo_computo, inicio, alto);                                                //Calcula el tiempo entre los eventos
        hipEventDestroy(inicio); hipEventDestroy(alto);                                                   // Se liberan los espacios  de los eventos para poder medir de nuevo más tarde

        cout << "Tiempo de computo en n^2 threads para una matriz de "<< TDM2 << ": "<<tiempo_computo << "ms"<<endl;

        hipEventCreate(&inicio); hipEventCreate(&alto);
        hipEventRecord(inicio);
        MultiplicarMatricesSecuencial<<<1, 1>>>(matriz1_GPU, matriz2_GPU, matriz3_GPU, TDM2, pitch);
        hipEventRecord(alto);
        hipEventSynchronize(alto);
        hipEventElapsedTime(&tiempo_computo, inicio, alto);
        hipEventDestroy(inicio); hipEventDestroy(alto);

        cout << "Tiempo de computo en secuencia para una matriz de "<< TDM2 << ": "<< tiempo_computo << "ms\n" << endl;
    }
    /* ******************** Muestra la matriz de salda de CUDA
    float matriz_salida[TDM][TDM];
    hipMemcpy2D(matriz_salida, TDM * sizeof(float), matriz3_GPU, pitch, TDM * sizeof(float), TDM, hipMemcpyDeviceToHost); //Copiamos los datos de la matriz de la GPU a la CPU
    
    cout.precision(3);

    cout << "\nMatriz Multiplicada" << endl;
    for(int i = 0; i < TDM; i++){
        for(int j = 0; j < TDM; j++){
            cout << matriz_salida[i][j] << "\t";
        }
        cout << "\n";
    }
    //free(matriz1_host); free(matriz2_host);*/
    hipFree(matriz1_GPU); hipFree(matriz2_GPU), hipFree(matriz3_GPU);

    

    return 0;
    
}