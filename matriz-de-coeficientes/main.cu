
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
using namespace std;

#define TDB 1024  //Tamaño del bloque 
#define hy 0.34
#define hx 0.34
#define LT 1 //lado tuberia

__device__ double my_floor(double num) {
    if (num >= LLONG_MAX || num <= LLONG_MIN || num != num) {
        return num;
    }
    int n = (int)num;
    double d = (double)n;
    if (d == num || num >= 0)
        return d;
    else
        return d - 1;
}

__global__ void crearMalla(float *matriz, float *coeficientes__GPU, int nodos_x, int nodos_y){
    const unsigned int idx = threadIdx.x;
    const unsigned int i = my_floor(idx / nodos_x);
    const unsigned int j = idx % nodos_x;
    const unsigned int n = nodos_x * nodos_y; 
    int k = 0, l = 0, columna = 0;
    printf("thread:%i \n", idx);
    while(k < nodos_x && columna < n ){
      
        while(l < nodos_y){
                    
            if( k == i - 1 && l == j ) 
                *(matriz + (idx * n) + columna) = *(coeficientes__GPU + 0);
            else if( k == i + 1 && l == j ) 
                *(matriz + (idx * n) + columna) = *(coeficientes__GPU + 1);
            else if( k == i && l == j ) 
                *(matriz + (idx * n) + columna) = *(coeficientes__GPU + 2);    //
            else if( k == i && l == j - 1 ) 
                *(matriz + (idx * n) + columna) = *(coeficientes__GPU + 3);
            else if( k == i && l == j + 1 ) 
                *(matriz + (idx * n) + columna) = *(coeficientes__GPU + 4);
            else 
                *(matriz + (idx * n) + columna) = 0;
            
            //printf("%.2f  ", *(matriz + (idx * n) + columna));
            columna++;

            l++;
        }
        l = 0;

        if(k < nodos_x) k++;
        else k = 0;
    }

} 
__global__ void gaussSeidel(){

}

int main(){
    const unsigned int nodos_x = ceil( (float)LT / (float)hx );
    const unsigned int nodos_y = ceil( (float)LT / (float)hy );
    const unsigned int NDH =  nodos_x * nodos_y; 
    const unsigned int numero_bloques =  ceil( (float) NDH / (float) TDB );
    const unsigned int hilos_bloque = ceil( (float) NDH / (float) numero_bloques );

    cout << " Se lanzaran " << numero_bloques << "bloque(s) de " << hilos_bloque << " hilos cada uno. \n\n";

    float* coeficientes__HOST =  (float*) malloc(5);
    *(coeficientes__HOST + 0 )=  1/(pow(hx,2)); //(i-1, j)
    *(coeficientes__HOST + 1) =  1/(pow(hx,2)); //(i+1, j)
    *(coeficientes__HOST + 2) = -2 *( (1/(pow(hx,2))) + (1/(pow(hy,2)))); //(i, j)
    *(coeficientes__HOST + 3) =  1/(pow(hy,2)); //(i, j+1)
    *(coeficientes__HOST + 4) =  1/(pow(hy,2)); //(i, j+1)

    //for(unsigned x = 0 ; x < 5 ; ++x) cout << *(coeficientes__HOST + x) << "\n";
    

    float *malla_salida_device; hipMalloc((void**)&malla_salida_device, NDH * NDH * sizeof(float));

    float *coeficientes__GPU; hipMalloc((void**)&coeficientes__GPU, 5 * sizeof(float));

    hipMemcpy(coeficientes__GPU, coeficientes__HOST, 5 * sizeof(float), hipMemcpyHostToDevice);

    crearMalla<<<numero_bloques, hilos_bloque>>>(malla_salida_device, coeficientes__GPU, nodos_x, nodos_y);

    float *malla_salida_host = (float*) malloc(NDH * NDH * sizeof(float));
    hipMemcpy(malla_salida_host, malla_salida_device, NDH * NDH * sizeof(float), hipMemcpyDeviceToHost);
  
    hipFree(malla_salida_device); hipFree(coeficientes__GPU);
    
    cout.precision(2);

    cout << "\n Nodos x: " << nodos_x << ", Nodos y: " << nodos_y << ", .\n \n"; 
    for( int i = 0; i < NDH; ++i){
        for(int j = 0 ; j < NDH ; ++j)
            cout <<   *(malla_salida_host + (i * NDH) + j) << "\t";

        cout << "\n";
        }

    free(coeficientes__HOST); free(malla_salida_host); 

    return 0;
} 